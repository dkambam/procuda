#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU(void){
	printf("Hello from GPU! %d\n", threadIdx.x);
}

int main(void){
	printf("Hello! from CPU\n");
	
	helloFromGPU <<< 1,10 >>>();

	// error handling
	hipError_t res = hipDeviceReset(); // enumerated error-code type
	if(res == hipSuccess){  // 
		printf("success!\n");
	}

}
